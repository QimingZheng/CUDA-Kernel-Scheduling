#include "hip/hip_runtime.h"
#include <pthread.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

const int N = 1 << 20;

__global__ void kernel(float *x, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // while(true);
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        x[i] = sqrt(pow(3.14159,i));
    }
}

struct args{
    float *data;
    int id;
};

void *launch_kernel(void *arg)
{
    float *data;
    hipMalloc(&data, N * sizeof(float));
    hipStream_t stream;
    hipStreamCreate(&stream);
    // kernel<<<1, 1, 64, stream>>>(((struct args*)arg)->data, N);
    kernel<<<1, 1, 64, stream>>>(data, N);
    if (((struct args*)arg)->id % 2) {
        checkCudaErrors(hipDeviceReset());
        return NULL;
    } else {
        checkCudaErrors(hipStreamSynchronize(stream));
        checkCudaErrors(hipMemcpy(((struct args*)arg)->data, data, N*sizeof(float), hipMemcpyDeviceToDevice));
        return NULL;
    }
}

int main()
{
    const int num_threads = 8;

    pthread_t threads[num_threads];
    float *data[num_threads];
    for (int i = 0 ; i < num_threads; i++) {
        checkCudaErrors(hipMalloc(&data[i], N * sizeof(float)));
    }

    for (int i = 0; i < num_threads; i++) {
        struct args *arg = new args;
        arg->data = data[i];
        arg->id = i;
        if (pthread_create(&threads[i], NULL, launch_kernel, arg)) {
            fprintf(stderr, "Error creating thread\n");
            return 1;
        }
    }

    for (int i = 0; i < num_threads; i++) {
        if(pthread_join(threads[i], NULL)) {
            fprintf(stderr, "Error joining thread\n");
            return 2;
        }
    }

    // hipDeviceReset();

    return 0;
}
